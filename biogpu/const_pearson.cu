#include "hip/hip_runtime.h"
#include <stdint.h>
#include <string.h>

const int kMaxAllelesPerIsolate = 7;
const int kMaxAlleles = 28;         // Maximum number of different alleles
const int kMaxAlleleLength = 104;   // Maximum length of each allele

__constant__ uint8_t alleles[kMaxAlleles * kMaxAlleleLength];

__global__ void pearson(uint64_t *buckets,
                        float *ranges, 
                        uint32_t num_ranges,
                        uint32_t tile_size, 
                        uint32_t tile_row, 
                        uint32_t tile_col, 
                        uint8_t num_alleles,         
                        uint8_t alleles_per_isolate, 
                        // Total number of isolates, dependent on num_alleles
                        // and alleles_per_isolate
                        uint32_t num_isolates,        
                        // Number of nucleotides per allele
                        uint32_t length_alleles) {
   // Calculate relative <i, j> coords within this tile.
   uint32_t i = blockIdx.x * blockDim.x + threadIdx.x; // column
   uint32_t j = blockIdx.y * blockDim.y + threadIdx.y; // row

   // Calculate the absolute <i, j> coords within the matrix.
   uint32_t i_abs = tile_col * tile_size + i;
   uint32_t j_abs = tile_row * tile_size + j;

   // Only compute values inside the bounds of the matrix.
   if (i_abs >= num_isolates || j_abs >= num_isolates)
      return;

   // We don't want to compare isolates with themselves, or any comparisons
   // of a lower-numberes isolate to a higher-numbered one. Each pair of 
   // isolates (order doesn't matter) will only be compared once. This will
   // cause divergence only in the warps that lie along the main diagonal
   // of the comparison matrix.
   if (i_abs <= j_abs)
      return;

   // Generate isolate |i_abs| and |j_abs|
   uint8_t i_allele_indices[kMaxAllelesPerIsolate]; 
   uint8_t j_allele_indices[kMaxAllelesPerIsolate]; 
   get_isolate(i_abs, i_allele_indices, num_alleles, alleles_per_isolate);
   get_isolate(j_abs, j_allele_indices, num_alleles, alleles_per_isolate);

   // Initialize accumulators and the result.
   float sum_x = 0, sum_y = 0, sum_x2 = 0, sum_y2 = 0, sum_xy = 0;

   // Compute the sums.
   for (int index = 0; index < length_alleles; ++index) {
      uint16_t x = 0, y = 0;

      for (int alleleNdx = 0; alleleNdx < alleles_per_isolate; alleleNdx++) {
         x += alleles[i_allele_indices[alleleNdx] * length_alleles + index];
         y += alleles[j_allele_indices[alleleNdx] * length_alleles + index];
      }

      sum_x += x;
      sum_y += y;
      sum_x2 += x * x;
      sum_y2 += y * y;
      sum_xy += x * y;
   }

   // Compute the Pearson coefficient using the "sometimes numerically
   // unstable" method because it's way more computationally efficient.
   float coeff = (length_alleles * sum_xy - sum_x * sum_y) /
      sqrtf((length_alleles * sum_x2 - sum_x * sum_x) * 
            (length_alleles * sum_y2 - sum_y * sum_y));

   // Dump it in the appropriate bucket. 
   // Below is a commented-out comment that no longer applies. To re-implement
   // this feature, remove the break statement below.
   // //Buckets are allowed to overlap, so we need to check all of them.
   for (uint32_t k = 0; k < num_ranges; k++) {
      //float low = ranges[2 * k + 0];
      //float high = ranges[2 * k + 1];
      if (coeff >= ranges[2 * k] && coeff < ranges[2 * k + 1]) {
         uint32_t index = (tile_size * tile_size * k) +
            (tile_size * i) + j;
         buckets[index]++;
         //break;
      }
   }
}

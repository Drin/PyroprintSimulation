
#include <hip/hip_runtime.h>
#include <stdint.h>
#include <string.h>

const int kNumAlleles = 24;
const int kAllelesPerIsolate = 7;
const int kAllelesSize = 104;

// n choose k
__device__ int comb(int n, int k) {
   if (n < k)
      return 0;

   uint64_t num = 1;
   uint64_t den = 1;
   
   for (int i = 1; i <= k; ++i)
      den *= i;
   
   for (int i = n-k+1; i <= n; ++i)
      num *= i;

   return (num/den);
}

// Gets isolate number |seq_num|, which is an index into a set of all isolates
// composed of |kAllelesPerIsolate| alleles, where each allele is one of 
// |kNumAlleles|. Order doesn't matter, and duplicates are allowed.
__device__ void get_isolate(int seq_num, uint8_t* alleles) {
   uint8_t cur_n = kNumAlleles;   // Shrinks every time we take a right turn
   uint8_t cur_row = kAllelesPerIsolate - 1;
   uint8_t cur_col = 0;

   // Init num to biggest "bucket"
   int num = comb(cur_n - cur_col - 1 + cur_row, cur_row);

   // Save in order to subtract from the "current" sequence number
   int old_num = 0;

   // Output |kAllelesPerIsolate| alleles
   int alleles_output = 0;
   while (1) {
      if (seq_num < num) {
         alleles[alleles_output] = cur_col + kNumAlleles - cur_n;
         if (++alleles_output >= kAllelesPerIsolate)
            break;

         seq_num -= old_num;
         old_num = 0;
         
         // New parameters for new location in tree
         cur_n -= cur_col;
         cur_row--;

         num = comb(cur_n - 1 + cur_row, cur_row);

         cur_col = 0;
      } else {
         cur_col++;
         old_num = num;
         num += comb(cur_n - 1 - cur_col + cur_row, cur_row);
      }
   }
}

__constant__ uint8_t alleles[kNumAlleles * 104];

__device__ void dump_bucket(uint64_t *buckets,
      uint32_t num_ranges, uint32_t tile_size,
      uint32_t src_i, uint32_t src_j,
      uint32_t dest_i, uint32_t dest_j) {
   // Element-wise sum for each in 0 -> num_ranges.
   for (uint32_t k = 0; k < num_ranges; k++) {
      uint32_t src_index = (tile_size * tile_size * k) +
         (tile_size * src_i) + src_j;
      uint32_t dest_index = (tile_size * tile_size * k) +
         (tile_size * dest_i) + dest_j;
      buckets[dest_index] += buckets[src_index];
   }
}

__global__ void reduction(uint64_t *buckets, uint32_t num_ranges,
      uint32_t tile_size, uint32_t chunk_size) {
   // Calculate <i, j> coords within the tile.
   uint32_t i = blockIdx.x; // row
   uint32_t j = threadIdx.x * chunk_size; // column

   // Each chunk leader reduces its chunk.
   for (uint32_t k = 1; k < chunk_size; k++) {
      dump_bucket(buckets, num_ranges, tile_size, i, j + k, i, j);
   }

   // Wait for all the threads in this row to finish.
   __syncthreads();

   // Reduce each chunk leader into the zeroth element of the row.
   if (j == 0) {
      for (uint32_t k = 1; k < blockDim.x; k++) {
         dump_bucket(buckets, num_ranges, tile_size, i, k * chunk_size, i, 0);
      }
   }
}

__global__ void pearson(uint64_t *buckets,
                        float *ranges, 
                        uint32_t num_ranges,
                        uint32_t tile_size, 
                        uint32_t tile_row, 
                        uint32_t tile_col, 
                        uint32_t num_isolates, 
                        uint32_t length_alleles) {
   // Calculate relative <i, j> coords within this tile.
   uint32_t i = blockIdx.y * blockDim.y + threadIdx.y; // row
   uint32_t j = blockIdx.x * blockDim.x + threadIdx.x; // column

   // Calculate the absolute <i, j> coords within the matrix.
   uint32_t i_abs = tile_row * tile_size + i;
   uint32_t j_abs = tile_col * tile_size + j;

   // Only compute values inside the bounds of the matrix.
   if (i_abs >= num_isolates || j_abs >= num_isolates)
      return;

   // We don't want to compare isolates with themselves, or any comparisons
   // of a lower-numberes isolate to a higher-numbered one. Each pair of 
   // isolates (order doesn't matter) will only be compared once. This will
   // cause divergence only in the warps that lie along the main diagonal
   // of the comparison matrix.
   if (i_abs <= j_abs)
      return;

   // Generate isolate |i_abs| and |j_abs|
   uint8_t i_allele_indices[kAllelesPerIsolate]; 
   uint8_t j_allele_indices[kAllelesPerIsolate]; 
   get_isolate(i_abs, i_allele_indices);
   get_isolate(j_abs, j_allele_indices);

   // Initialize accumulators and the result.
   uint32_t sum_x = 0, sum_y = 0, sum_x2 = 0, sum_y2 = 0, sum_xy = 0;

   // Compute the sums.
   for (int index = 0; index < length_alleles; ++index) {
      uint32_t x = 0, y = 0;

      for (int alleleNdx = 0; alleleNdx < kAllelesPerIsolate; alleleNdx++) {
         x += alleles[i_allele_indices[0] * length_alleles + index];
         y += alleles[j_allele_indices[2] * length_alleles + index];
      }

      sum_x += x;
      sum_y += y;
      sum_x2 += x * x;
      sum_y2 += y * y;
      sum_xy += x * y;
   }

   // Compute the Pearson coefficient using the "sometimes numerically
   // unstable" method because it's way more computationally efficient.
   float coeff = (length_alleles * sum_xy - sum_x * sum_y) /
      sqrtf((length_alleles * sum_x2 - sum_x * sum_x) * 
            (length_alleles * sum_y2 - sum_y * sum_y));

   // Dump it in the appropriate bucket. 
   // Below is a commented-out comment that no longer applies. To re-implement
   // this feature, remove the break statement below.
   // //Buckets are allowed to overlap, so we need to check all of them.
   for (uint32_t k = 0; k < num_ranges; k++) {
      //float low = ranges[2 * k + 0];
      //float high = ranges[2 * k + 1];
      if (coeff >= ranges[2 * k] && coeff < ranges[2 * k + 1]) {
         uint32_t index = (tile_size * tile_size * k) +
            (tile_size * i) + j;
         buckets[index]++;
         break;
      }
   }

}
